#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>
#include <vector>

namespace votenet {

int const threadsPerBlock = sizeof(unsigned long long) * 8;

template <typename integer>
constexpr __host__ __device__ inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template <typename T>
__device__ inline bool devIoU(T const* const a, T const* const b, const float threshold) {
  T left = max(a[0], b[0]), right = min(a[3], b[3]);
  T back = max(a[1], b[1]), front = min(a[4], b[4]);
  T top = max(a[2], b[2]), bottom = min(a[5], b[5]);
  T width = max(right - left, (T)0), depth = max(front - back, (T)0), height = max(bottom - top, (T)0);
  T interV = width * depth * height;
  T Va = (a[3] - a[0]) * (a[4] - a[1]) * (a[5] - a[2]);
  T Vb = (b[3] - b[0]) * (b[4] - b[1]) * (b[5] - b[2]);
  return (interV / (Va + Vb - interV)) > threshold;
}

template <typename T>
__global__ void nms_3d_kernel(
    int n_boxes,
    float iou_threshold,
    const T* dev_boxes,
    unsigned long long* dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  if (row_start > col_start) return;

  const int row_size =
      min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ T block_boxes[threadsPerBlock * 6];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 6 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 0];
    block_boxes[threadIdx.x * 6 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 1];
    block_boxes[threadIdx.x * 6 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 2];
    block_boxes[threadIdx.x * 6 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 3];
    block_boxes[threadIdx.x * 6 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 4];
    block_boxes[threadIdx.x * 6 + 5] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 5];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const T* cur_box = dev_boxes + cur_box_idx * 6;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU<T>(cur_box, block_boxes + i * 6, iou_threshold)) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = ceil_div(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

at::Tensor nms_3d_cuda(const at::Tensor& dets,
    const at::Tensor& scores,
    const float iou_threshold) {
  TORCH_CHECK(dets.is_cuda(), "dets must be a CUDA tensor");
  TORCH_CHECK(scores.is_cuda(), "scores must be a CUDA tensor");

  TORCH_CHECK(
      dets.dim() == 2, "boxes should be a 2d tensor, got ", dets.dim(), "D");
  TORCH_CHECK(
      dets.size(1) == 6,
      "boxes should have 6 elements in dimension 1, got ",
      dets.size(1));
  TORCH_CHECK(
      scores.dim() == 1,
      "scores should be a 1d tensor, got ",
      scores.dim(),
      "D");
  TORCH_CHECK(
      dets.size(0) == scores.size(0),
      "boxes and scores should have same number of elements in ",
      "dimension 0, got ",
      dets.size(0),
      " and ",
      scores.size(0))

#if defined(WITH_CUDA) || defined(WITH_HIP)
  at::cuda::CUDAGuard device_guard(dets.device());
#else
  TORCH_CHECK(false, "Not compiled with GPU support");
#endif

  if (dets.numel() == 0) {
    return at::empty({0}, dets.options().dtype(at::kLong));
  }

  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto dets_sorted = dets.index_select(0, order_t).contiguous();

  int dets_num = dets.size(0);

  const int col_blocks = ceil_div(dets_num, threadsPerBlock);

  at::Tensor mask =
      at::empty({dets_num * col_blocks}, dets.options().dtype(at::kLong));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      dets_sorted.scalar_type(), "nms_kernel_cuda", [&] {
        nms_3d_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            dets_num,
            iou_threshold,
            dets_sorted.data_ptr<scalar_t>(),
            (unsigned long long*)mask.data_ptr<int64_t>());
      });

  at::Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host = (unsigned long long*)mask_cpu.data_ptr<int64_t>();

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep =
      at::empty({dets_num}, dets.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data_ptr<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < dets_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  AT_CUDA_CHECK(hipGetLastError());
  return order_t.index(
      {keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep)
           .to(order_t.device(), keep.scalar_type())});
}

} // namespace votenet
